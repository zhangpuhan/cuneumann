#include "hip/hip_runtime.h"
//
// Created by Puhan Zhang on 1/12/21.
//

#include <hip/hip_runtime_api.h>
#include "model.cuh"
#include "util.h"
#include <stdio.h>
#include <assert.h>
#include "hipsolver.h"
#include "hipsparse.h"
#include "hip/hip_complex.h"
#include "kernel.cuh"
#include "hipblas.h"


Square_Lattice::Square_Lattice(int linear_size) {

    L = linear_size;
    Ns = L * L;
    dim = Ns;

    site = new Site[Ns];

    hipMallocManaged(&Hamiltonian, dim * dim  * sizeof(hipDoubleComplex));

    hipMallocManaged(&Density_Mat, dim * dim  * sizeof(hipDoubleComplex));
    hipMallocManaged(&actor, dim * dim  * sizeof(hipDoubleComplex));
    hipMallocManaged(&neighboring, 4 * dim * sizeof(int));
    hipMallocManaged(&onsite_V, dim  * sizeof(double));
    init_lattice();

    time = 0;

}

void Square_Lattice::init_lattice() {

    for (int x = 0; x < L; x++)
        for (int y = 0; y < L; y++) {
            int idx = index(x, y);
            site[idx].idx = idx;
            site[idx].x = x;
            site[idx].y = y;

            site[idx].sgn = ((x + y) % 2 == 0) ? +1 : -1;

        }

    for (int i = 0; i < Ns; i++) {
        int j;
        int x = site[i].x;
        int y = site[i].y;

        j = index(mod(x + 1, L), y);
        site[i].nn1[0] = &site[j];
        neighboring[4 * i] = site[j].idx;

        j = index(mod(x - 1, L), y);
        site[i].nn1[1] = &site[j];
        neighboring[4 * i + 1] = site[j].idx;

        j = index(x, mod(y + 1, L));
        site[i].nn1[2] = &site[j];
        neighboring[4 * i + 2] = site[j].idx;


        j = index(x, mod(y - 1, L));
        site[i].nn1[3] = &site[j];
        neighboring[4 * i + 3] = site[j].idx;

    }
}

void Square_Lattice::build_Hamiltonian() {
    for(int i = 0; i < Ns; i++) {
        for(auto & k : site[i].nn1) {
            int j = k->idx;
            Hamiltonian[i * dim + j].x = t1;
        }
    }

    for(int i = 0; i < Ns; i++) {
        Hamiltonian[i * dim + i].x = onsite_V[i];
    }
}

void Square_Lattice::integrate_EOM_RK4(double dt) {

    hipDoubleComplex *D2 = nullptr;
    hipDoubleComplex *KD_sum = nullptr;
    hipMallocManaged(&D2, dim * dim * sizeof(hipDoubleComplex));
    cuZinitialize<<<128, 128>>>(D2, dim);
    hipDeviceSynchronize();
    hipMallocManaged(&KD_sum, dim * dim * sizeof(hipDoubleComplex));
    cuZinitialize<<<128, 128>>>(KD_sum, dim);
    hipDeviceSynchronize();
    // hipMemset(D2, {0.0, 0.0}, dim * dim * sizeof(hipDoubleComplex));
    // hipMemset(KD_sum, {0.0, 0.0}, dim * dim * sizeof(hipDoubleComplex));
    // hipDeviceSynchronize();

    hipDoubleComplex n_tot = {0.0, 0.0};
    for (int i = 0; i < dim; ++i) {
        n_tot = hipCadd(KD_sum[i * dim + i], n_tot);
    }
    // std::cout <<  "kd_trace = " << n_tot.x << " + " << n_tot.y << "I"<< std::endl;

    // step 1
    step(KD_sum, D2, dt, 1);
    
    // step 2
    cuZbuild_Hamiltonian<<<128, 128>>>(Hamiltonian, dim, onsite_V, neighboring, t1, N_nn1);
    hipDeviceSynchronize();

    step(KD_sum, D2, dt, 2);

    // step 3
    cuZbuild_Hamiltonian<<<128, 128>>>(Hamiltonian, dim, onsite_V, neighboring, t1, N_nn1);
    hipDeviceSynchronize();

    step(KD_sum, D2, dt, 3);


    // step 4
    cuZbuild_Hamiltonian<<<128, 128>>>(Hamiltonian, dim, onsite_V, neighboring, t1, N_nn1);
    hipDeviceSynchronize();
    step(KD_sum, D2, dt, 4);
        
    // ------- RK4: sum all steps: ------------
    
    // Density_Mat = D + KD_sum;
    
    // // compute the system Hamiltonian, R, Delta:

    hipblasHandle_t handle_add;
    hipblasCreate(&handle_add);
    hipDoubleComplex alpha           = {1.0, 0.0};
    hipDoubleComplex beta            = {1.0, 0.0};

    hipblasZgeam(handle_add,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        dim, dim,
        &alpha,
        KD_sum, dim,
        &beta,
        Density_Mat, dim,
        Density_Mat, dim);
    
    hipDeviceSynchronize();
    hipblasDestroy(handle_add);
    n_tot = {0.0, 0.0};
    for (int i = 0; i < dim; ++i) {
        n_tot = hipCadd(KD_sum[i * dim + i], n_tot);
    }
    // std::cout <<  "time = "<< time << ", kd_trace = " << n_tot.x << " + " << n_tot.y << "I"<< std::endl;
    // printf("KD_sum = %7.6lf + %7.6lfi\n", KD_sum[dim].x, KD_sum[dim].y);
    // printf("1*************\n");
    // for (int i = 0; i < dim; i++) {
    //     for (int j = 0; j < dim; j++) {
    //             printf("%7.6lf + %7.6lf*I   ", KD_sum[i * dim + j].x, KD_sum[i * dim + j].y);
    //     }
    //     printf ("\n");
    // }
    // printf("1*************\n");

    // printf("*************\n");
    // for (int i = 0; i < dim; i++) {
    //     for (int j = 0; j < dim; j++) {
    //             printf("%7.6lf + %7.6lf*I   ", Density_Mat[i * dim + j].x, Density_Mat[i * dim + j].y);
    //     }
    //     printf ("\n");
    // }

    cuZbuild_Hamiltonian<<<128, 128>>>(Hamiltonian, dim, onsite_V, neighboring, t1, N_nn1);
    hipDeviceSynchronize();


    hipFree(KD_sum);
    hipFree(D2);
}

void Square_Lattice::compute_fermi_level(double *eigE) {

    double x1 = eigE[0];
    double x2 = eigE[dim - 1];

    int max_bisection = 500;
    double eps_bisection = 1.e-12;

    int iter = 0;
    while(iter < max_bisection || fabs(x2 - x1) > eps_bisection) {

        double xm = 0.5 * (x1 + x2);
        double density = 0;
        for(int i=0; i<dim; i++) {
            density += fermi_density(eigE[i], kT, xm);
        }
        density /= ((double) dim);

        if(density <= filling) x1 = xm;
        else x2 = xm;

        iter++;
    }

    mu = 0.5*(x1 + x2);
}

// void Square_Lattice::compute_density_matrix() {

//     hipsolverHandle_t cusolverH = nullptr;
//     hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
//     int *devInfo = nullptr;
//     int lwork = 0;
//     hipDoubleComplex *d_work = nullptr;

//     double *eigval = nullptr;
//     hipDoubleComplex *eigvec = nullptr;

//     hipMallocManaged(&eigval, dim * sizeof(double));
//     hipMallocManaged(&eigvec, dim * dim * sizeof(hipDoubleComplex));
//     hipMallocManaged(&devInfo, sizeof(int));

//     hipMemcpy(eigvec, Hamiltonian, dim * dim * sizeof(hipDoubleComplex), hipMemcpyDefault);
//     hipDeviceSynchronize();

//     cusolver_status = hipsolverDnCreate(&cusolverH);
//     hipDeviceSynchronize();
//     assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

//     hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; 
//     hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

//     cusolver_status = hipsolverDnZheevd_bufferSize(cusolverH, jobz, uplo, dim, eigvec, dim, eigval, &lwork);
//     hipDeviceSynchronize();
//     assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

//     hipMallocManaged(&d_work, lwork * sizeof(hipDoubleComplex));
//     cusolver_status = hipsolverDnZheevd(cusolverH, jobz, uplo, dim, eigvec, dim, eigval, d_work, lwork, devInfo);
//     hipDeviceSynchronize();
//     assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
//     assert(0 == *devInfo);

//     std::ofstream fs("eig.dat");
//     for(int r=0; r<dim; r++) {
//         fs << eigval[r] << std::endl;
//     }
//     fs.close();

//     compute_fermi_level(eigval);
//     hipDoubleComplex *fd_factor;
//     hipMallocManaged(&fd_factor, dim * sizeof(hipDoubleComplex));
//     for(int i=0; i<dim; i++) {
//         fd_factor[i].x = fermi_density(eigval[i], kT, mu);
//     } 
//     hipDoubleComplex sum;
//     for(int a = 0; a < dim; a++) {
//         for(int b = a; b < dim; b++) {

//             sum = {0.0, 0.0};
//             for(int m = 0; m < dim; m++) {
//                 sum = hipCadd(hipCmul(fd_factor[m], hipCmul(hipConj(eigvec[m * dim + a]), eigvec[m * dim + b])), sum);
//             }
//             Density_Mat[a * dim + b] = sum;
//             if(a != b) Density_Mat[b * dim + a] = hipConj(sum);
//         }
//     }

//     hipFree(eigval);
//     hipFree(eigvec);
//     hipFree(d_work);
//     hipFree(devInfo);
//     hipFree(fd_factor);
//     hipsolverDnDestroy(cusolverH);
// }

void Square_Lattice::compute_density_matrix() {
    for (int i = 0; i < dim; ++i) {
        if (i % 2 == 0) {
            Density_Mat[i * dim + i].x = 1.0;
            Density_Mat[i * dim + i].y = 0.0;
        }
    }
}

// void Square_Lattice::save_configuration(std::string const filename) {

//     std::ofstream fs;

//     fs.open(filename.c_str(), std::ios::out);
//     fs.precision(12);

//     for(int i=0; i<Ns; i++) {

//         fs << real(Density_Mat(2 * i, 2 * i) + Density_Mat(2 * i + 1, 2 * i + 1)) << '\t';
//         fs << std::endl;
//     }
//     fs.close();
// }

void Square_Lattice::step(hipDoubleComplex *KD_sum, hipDoubleComplex *D2, double dt, int step) {
    // convert Hamiltonian to CSR sparse matrix
    hipsparseHandle_t handle_convert = nullptr;
    hipsparseDirection_t direction = HIPSPARSE_DIRECTION_ROW;
    hipsparseMatDescr_t Htype;
    hipsparseCreateMatDescr(&Htype);
    hipsparseSetMatType(Htype, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(Htype, HIPSPARSE_INDEX_BASE_ZERO);

    int nnz = 0;
    int *H_offset;
    
    hipMallocManaged(&H_offset, dim * sizeof(int));
    hipsparseCreate(&handle_convert);
    hipsparseZnnz(handle_convert, direction, dim, dim, Htype, Hamiltonian, dim, H_offset, &nnz);
    hipDeviceSynchronize();


    hipsparseHandle_t handle_csr = nullptr;
    hipsparseCreate(&handle_csr);
    hipDoubleComplex *H_val;
    int *H_row, *H_col;

    hipMallocManaged(&H_val, nnz * sizeof(hipDoubleComplex));
    hipMallocManaged(&H_row, (dim + 1) * sizeof(int));
    hipMallocManaged(&H_col, nnz * sizeof(int));


    // CSR format of H created
    hipsparseZdense2csr(handle_csr, dim, dim, Htype, Hamiltonian, dim, H_offset, H_val, H_row, H_col);
    hipDeviceSynchronize();

    // step 1
    // ------- RK4 step-1: ----------------
    
    // KD = -_I * dt * ( H * D - D * H );
    
    // D2 = D + 0.5 * KD;
    // KD_sum = KD / 6.;

    hipsparseHandle_t     handle_mm = nullptr;
    hipsparseSpMatDescr_t H;
    hipsparseDnMatDescr_t D, matT;
    void*                dBuffer = nullptr;
    size_t               bufferSize = 0;
    hipDoubleComplex      *KD;
    hipMallocManaged(&KD, dim * dim * sizeof(hipDoubleComplex));

    hipsparseCreate(&handle_mm);

    // create sparse H, D, KD in mat format
    hipsparseCreateCsr(&H, dim, dim, nnz, H_row, H_col, H_val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);
    if (step == 1) {
        hipsparseCreateDnMat(&D, dim, dim, dim, Density_Mat, HIP_C_64F, HIPSPARSE_ORDER_ROW);
    }
    else {
        hipsparseCreateDnMat(&D, dim, dim, dim, D2, HIP_C_64F, HIPSPARSE_ORDER_ROW);
    }
    hipsparseCreateDnMat(&matT, dim, dim, dim, KD, HIP_C_64F, HIPSPARSE_ORDER_ROW);

    // allocation buffer
    hipDoubleComplex alpha           = {1.0, 0.0};
    hipDoubleComplex beta            = {1.0, 0.0};
    hipsparseSpMM_bufferSize(handle_mm, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, H, D, &beta, matT, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMallocManaged(&dBuffer, bufferSize);

    // execute SpMM
    // compute KD = DH, H*D*
    hipsparseSpMM(handle_mm, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE, &alpha, H, D, &beta, matT, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
    hipDeviceSynchronize();


    hipDoubleComplex energy = {0.0, 0.0};
    if (step == 1) {
        for (int ti = 0; ti < dim; ++ti) {
            energy = hipCadd(energy, KD[ti * dim + ti]);
        }
        printf("energy = %7.6lf + %7.6lfi\n", energy.x, energy.y);
    }

    // if (step == 1) {
        // printf("step%d, D2*************\n", step);
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%7.6lf + %7.6lf*I   ", D2[i * dim + j].x, D2[i * dim + j].y);
        //     }
        //     printf ("\n");
        // }
    // }




    // conjugate KD
    hipblasHandle_t handle_trans;
    hipblasCreate(&handle_trans);
    alpha = {1.0, 0.0};
    beta = {0.0, 0.0};

    hipblasZgeam(handle_trans, HIPBLAS_OP_C, HIPBLAS_OP_N, dim, dim, &alpha, KD, dim, &beta, actor, dim, actor, dim);
    
    hipMemcpy(KD, actor, dim * dim * sizeof(hipDoubleComplex), hipMemcpyDefault);
    hipDeviceSynchronize();
    hipblasDestroy(handle_trans);
    cuZinitialize<<<128, 128>>>(actor, dim);
    hipDeviceSynchronize();

    // if (step == 1) {
        // printf("step%d, KD=DH*************\n", step);
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%6.5lf + %6.5lf*I   ", KD[i * dim + j].x, KD[i * dim + j].y);
        //     }
        //     printf ("\n");
        // }
    // }

    // if (step == 1) {
        // printf("step%d, Density_Mat*************\n", step);
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%5.3lf+1i*%5.3lf ", Density_Mat[i * dim + j].x, Density_Mat[i * dim + j].y);
        //     }
        //     printf (";\n");
        // }
    // }

    // compute KD = -Idt HD + Idt 
    alpha = {0.0, -dt};
    beta = {0.0, dt};
    hipsparseSpMM_bufferSize(handle_mm, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, H, D, &beta, matT, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMallocManaged(&dBuffer, bufferSize);
    hipsparseSpMM(handle_mm, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, H, D, &beta, matT, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
    hipDeviceSynchronize();

    // if (step == 1) {
        // printf("step%d, totalKD*************\n", step);
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%7.6lf + %7.6lf*I   ", KD[i * dim + j].x, KD[i * dim + j].y);
        //     }
        //     printf ("\n");
        // }
    // }


    // compute D2 = D + 0.5 * KD
    hipblasHandle_t handle_add;
    hipblasCreate(&handle_add);
    alpha           = {1.0, 0.0};
    beta            = {0.5, 0.0};

    hipblasZgeam(handle_trans, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, &alpha, Density_Mat, dim, &beta, KD, dim, D2, dim);
    hipDeviceSynchronize();
    hipblasDestroy(handle_add);



    // if (step == 1) {
        // printf("step%d, D2*************\n", step);
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%7.6lf + %7.6lf*I   ", D2[i * dim + j].x, D2[i * dim + j].y);
        //     }
        //     printf ("\n");
        // }
    // }

    // compute KD_sum += KD / 6 or KD / 3;
    if (step == 2 || step == 3) {
        alpha = {1.0 / 3.0, 0.0};

    }
    else {
        alpha = {1.0 / 6.0, 0.0};
    }
    hipblasHandle_t handle_trans_2;
    hipblasCreate(&handle_trans_2);
    beta = {1.0, 0.0};

    hipblasZgeam(handle_trans, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, &alpha, KD, dim, &beta, KD_sum, dim, KD_sum, dim);
    hipDeviceSynchronize();
    hipblasDestroy(handle_trans_2);

    // if (step == 4) {
    //     printf("KD_sum*************\n");
    //     for (int i = 0; i < dim; i++) {
    //         for (int j = 0; j < dim; j++) {
    //                 printf("%7.6lf + %7.6lf*I   ", KD_sum[i * dim + j].x, KD_sum[i * dim + j].y);
    //         }
    //         printf ("\n");
    //     }
    // }

    hipsparseDestroySpMat(H);
    hipsparseDestroyDnMat(D);
    hipsparseDestroyDnMat(matT);
    hipsparseDestroy(handle_mm);
    hipsparseDestroy(handle_convert);
    hipsparseDestroy(handle_csr);
    hipFree(H_offset);
    hipFree(H_val);
    hipFree(H_row);
    hipFree(H_col);
    hipFree(dBuffer);
    hipFree(KD);
}

hipDoubleComplex Square_Lattice::calculate_trace() {
    int blocksPerGrid = min(32, (dim + 256 - 1) / 256);
    hipDoubleComplex *trace;
    hipMallocManaged(&trace, blocksPerGrid * sizeof(hipDoubleComplex));
    cuZtrace<<<256, 256>>>(Density_Mat, 4, trace);
    hipDeviceSynchronize();
    hipDoubleComplex result = {0.0, 0.0};
    for (int i = 0; i < blocksPerGrid; ++i) {
        result = hipCadd(result, trace[i]);
    }
    return result;
}

void Square_Lattice::init_quenched_disorder(double W) {

    std::random_device seed;

    RNG rng = RNG(seed());

    std::uniform_real_distribution<double> rd(-W, W);

    for(int i=0; i<Ns; i++) onsite_V[i] = rd(rng);
}

void Square_Lattice::simulate_dynamics(int max_steps, double dt, double W) {
    init_quenched_disorder(W);
    cuZinitialize<<<128, 128>>>(Hamiltonian, dim);
    hipDeviceSynchronize();
    cuZbuild_Hamiltonian<<<128, 128>>>(Hamiltonian, dim, onsite_V, neighboring, t1, N_nn1);
    hipDeviceSynchronize();

    cuZinitialize<<<128, 128>>>(Density_Mat, dim);
    hipDeviceSynchronize();

    compute_density_matrix();




    time = 0;
    for(int i = 0; i < 1000; i++) {

        std::cout << "i = " << i << std::endl;
        
        // printf("D*************\n");
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%7.6lf + %7.6lf*i   ", Density_Mat[i * dim + j].x, Density_Mat[i * dim + j].y);
        //     }
        //     printf (";\n");
        // }

        // printf("H*************\n");
        // for (int i = 0; i < dim; i++) {
        //     for (int j = 0; j < dim; j++) {
        //             printf("%7.6lf + %7.6lf*i   ", Hamiltonian[i * dim + j].x, Hamiltonian[i * dim + j].y);
        //     }
        //     printf (";\n");
        // }
        // printf("2*************\n");
        hipDoubleComplex n_tot = {0.0, 0.0};
        for (int i = 0; i < dim; ++i) {
            n_tot = hipCadd(Density_Mat[i * dim + i], n_tot);
        }
        printf("time = %5.3lf, trace = %7.6lf + %7.6lfi\n", time, n_tot.x, n_tot.y);
        printf("D0 = %7.6lf + %7.6lfi\n", Density_Mat[0].x, Density_Mat[0].y);
        printf("D1 = %7.6lf + %7.6lfi\n", Density_Mat[dim + 1].x, Density_Mat[dim + 1].y);
        printf("D2 = %7.6lf + %7.6lfi\n", Density_Mat[2 * dim + 2].x, Density_Mat[2 * dim + 2].y);

        integrate_EOM_RK4(dt);
        time += dt; 
    }
    hipFree(Hamiltonian);
    hipFree(Density_Mat);
    hipFree(onsite_V);
    hipFree(neighboring);
    hipFree(actor);
}
